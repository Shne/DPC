#include "hip/hip_runtime.h"
#include "FinalPassKernel.h"
#include <stdio.h>

#define E (2.7182818284f)


__global__
void finalPassKernel(int height, int width, HitInfo* dev_scatteringMPs, HitInfo* hi) {
	
	// int i = blockIdx.x*blockDim.x + threadIdx.x;
	// int j = blockIdx.y*blockDim.y + threadIdx.y;
	// HitInfo* hi = dev_eyeMPs[j*width + i];

	if(hi == NULL) return;

	const float my = 1.3;
	const float sigmaS = 2.6		* 60; // TranslucentMaterialScale;
	const float sigmaA = 0.0041 	* 60; //TranslucentMaterialScale;
	const float sigmaT = sigmaS + sigmaA;
	// const float alpha = sigmaS / sigmaT;
	const float sigmaTR = sqrt(3.0*sigmaA*sigmaT);
	const float lu = 1.0/sigmaT;
	const float Fdr =  -1.440/(my*my) + 0.710*my + 0.668 + 0.0636*my;
	const float Fdt = 1.0 - Fdr;
	const float A = (1 + Fdr) / (1 - Fdr);
	const float zr = lu;
	const float zv = lu*(1.0 + 4.0/(3.0*A));

	// MULTIPLE SCATTER
	// int scatteringMPsSize = scatteringMPs.size();
	// for(int i=0; i<scatteringMPsSize; i++) {
	// 	HitInfo* sHI = scatteringMPs[i];
	HitInfo sHI = dev_scatteringMPs[blockIdx.x*blockDim.x + threadIdx.x];

	
	float r2 = (hi->P - sHI.P).length2();
	float dr = sqrt(r2+zr*zr);
	float dv = sqrt(r2+zv*zv);
	float C1 = zr * (sigmaTR + 1.0f/dr);
	float C2 = zv * (sigmaTR + 1.0f/dv);

	float dMoOverAlphaPhi = 1.0f/(4.0f*PI) * (C1*(pow(E,-sigmaTR*dr)/dr*dr) + C2*(pow(E,-sigmaTR*dv)/dv*dv));
	Vector3 MoP = Fdt * dMoOverAlphaPhi * sHI.flux * sHI.r2 * PI;
	hi->flux += MoP;


	
}

extern "C" __host__
HitInfo* finalPass(Image* img, HitInfo* scatteringMPs, int scatteringMPsSize, HitInfo* measureHIArray, Camera* cam) {
	int width = img->width();
	int height = img->height();

	static HitInfo *dev_scatteringMPs, *dev_eyeMPs;
	hipMalloc((void**)&dev_scatteringMPs, scatteringMPsSize*sizeof(HitInfo));
	hipMalloc((void**)&dev_eyeMPs, width*height*sizeof(HitInfo));
	// hipMalloc((void**)&dev_hi, sizeof(HitInfo*));
	// hipMalloc((void**)&dev_eyeMPs, eyeMPs->size()*sizeof(HitInfo*));

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	hipMemcpy( dev_scatteringMPs, scatteringMPs, scatteringMPsSize*sizeof(HitInfo), hipMemcpyHostToDevice );
	hipMemcpy( dev_eyeMPs, measureHIArray, width*height*sizeof(HitInfo), hipMemcpyHostToDevice );

	// Kernel block dimensions
	// const dim3 blockDim(16,16);

	for (int j = 0; j < img->height(); ++j) {
		for (int i = 0; i < img->width(); ++i) {
			HitInfo hi = measureHIArray[j*img->width() + i];
			if(hi.t == 0.0f) continue;
			// finalPassKernel<<<dim3(width/blockDim.x, height/blockDim.y), blockDim>>>(height, width, dev_scatteringMPs, hi);
			finalPassKernel<<<64, 256>>>(height, width, dev_scatteringMPs, &hi);

			// Check for errors
			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
		}
	}



	HitInfo* result = new HitInfo[width*height];
	hipMemcpy(result, dev_eyeMPs, width*height*sizeof(HitInfo), hipMemcpyDeviceToHost );

	return result;
}

