#include "hip/hip_runtime.h"
#include "FinalPassKernel.h"
#include <stdio.h>
// #include "hip/hip_runtime_api.h"

#define E (2.7182818284f)

using namespace std;


__global__
void finalPassKernel(const int height, const int width, const HitInfo* dev_scatteringMPs, const int scatteringMPsSize, const HitInfo* hiArray, const int hiIndex, Vector3 *scatteringMPsFlux, const float translucentMaterialScale) {
	
	// int i = blockIdx.x*blockDim.x + threadIdx.x;
	// int j = blockIdx.y*blockDim.y + threadIdx.y;
	// HitInfo* hi = dev_eyeMPs[j*width + i];
	const HitInfo hi = hiArray[hiIndex];

	// if(hi == NULL) return;

	const float my = 1.3;
	const float sigmaS = 2.6f		* translucentMaterialScale;
	const float sigmaA = 0.0041f 	* translucentMaterialScale;
	const float sigmaT = sigmaS + sigmaA;
	// const float alpha = sigmaS / sigmaT;
	const float sigmaTR = sqrt(3.0f*sigmaA*sigmaT);
	const float lu = 1.0f/sigmaT;
	const float Fdr =  -1.440f/(my*my) + 0.710f*my + 0.668f + 0.0636f*my;
	const float Fdt = 1.0f - Fdr;
	const float A = (1 + Fdr) / (1 - Fdr);
	const float zr = lu;
	const float zv = lu*(1.0f + 4.0f/(3.0f*A));


/*
	//PRECOMPUTED VALUES
	// const float my = 1.3;
	// const float sigmaS = 156;
	// const float sigmaA = 0.246;
	// const float sigmaT = 156.246;
	const float sigmaTR = 10.73822834549535922654;
	// const float lu = 0.00640016384419441138;
	// const float Fdr = 0.82160899408284023669;
	const float Fdt = 0.17839100591715976331;
	// const float A = 10.21132755386080971017;
	const float zr = 0.00640016384419441138;
	const float zv = 0.00723585849283319265;

*/


	// MULTIPLE SCATTER
	// int scatteringMPsSize = scatteringMPs.size();
	// for(int i=0; i<scatteringMPsSize; i++) {
	// 	HitInfo* sHI = scatteringMPs[i];
	const int scatteringMPsIndex = blockIdx.x*blockDim.x + threadIdx.x;
	const HitInfo sHI = dev_scatteringMPs[scatteringMPsIndex];

	
	const float r2 = (hi.P - sHI.P).length2();
	const float dr = sqrt(r2+zr*zr);
	const float dv = sqrt(r2+zv*zv);
	const float C1 = zr * (sigmaTR + 1.0f/dr);
	const float C2 = zv * (sigmaTR + 1.0f/dv);

	const float dMoOverAlphaPhi = 1.0f/(4.0f*PI) * (C1*(pow(E,-sigmaTR*dr)/dr*dr) + C2*(pow(E,-sigmaTR*dv)/dv*dv));
	const Vector3 MoP = Fdt * dMoOverAlphaPhi * sHI.flux * sHI.r2 * PI;

	scatteringMPsFlux[scatteringMPsIndex] = MoP;
	// hi.flux += MoP;


	
}

extern "C" __host__
HitInfo* finalPass(const int width, const int height, const HitInfo* scatteringMPs, const int scatteringMPsSize, HitInfo* measureHIArray, const float translucentMaterialScale) {
	// hipProfilerStart();

	static HitInfo *dev_scatteringMPs, *dev_eyeMPs;
	static Vector3 *scatteringMPsFlux;
	hipMalloc((void**)&dev_scatteringMPs, scatteringMPsSize*sizeof(HitInfo));
	hipMalloc((void**)&scatteringMPsFlux, scatteringMPsSize*sizeof(Vector3));
	hipMalloc((void**)&dev_eyeMPs, width*height*sizeof(HitInfo));
	

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	hipMemcpy( dev_scatteringMPs, scatteringMPs, scatteringMPsSize*sizeof(HitInfo), hipMemcpyHostToDevice );
	hipMemcpy( dev_eyeMPs, measureHIArray, width*height*sizeof(HitInfo), hipMemcpyHostToDevice );

	Vector3 *perPixelFlux = new Vector3[scatteringMPsSize];


	for (int j = 0; j < height; ++j) {
		for (int i = 0; i < width; ++i) {
			HitInfo hi = measureHIArray[j*width + i];
			if(hi.material == NULL) continue;
			// finalPassKernel<<<dim3(width/blockDim.x, height/blockDim.y), blockDim>>>(height, width, dev_scatteringMPs, hi);
			// std::cout << 'H'; fflush(stdout);
			// finalPassKernel<<<dim3(width/blockDim.x, height/blockDim.y), blockDim>>>(height, width, dev_scatteringMPs, dev_eyeMPs, j*img->width() + i);

			dim3 dimBlock(256);
			dim3 dimGrid = scatteringMPsSize/dimBlock.x;

			finalPassKernel<<<dimGrid, dimBlock>>>(height, width, dev_scatteringMPs, scatteringMPsSize, dev_eyeMPs, j*width + i, scatteringMPsFlux, translucentMaterialScale);
			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
			// cout << "after kernel" << endl;
			hipMemcpy(perPixelFlux, scatteringMPsFlux, scatteringMPsSize*sizeof(Vector3), hipMemcpyDeviceToHost );
			Vector3 flux;
			for(int _i = 0; _i < scatteringMPsSize; _i++) {
				flux += perPixelFlux[_i];
				// std::cout << perPixelFlux[_i];
			}
			// cout << "after sum" << endl;
			// cout << measureHIArray[j*width+i].flux << " ";
			measureHIArray[j*width+i].flux = flux;
			// cout << measureHIArray[j*width+i].flux << endl;

			// Check for errors
			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
		}
		printf("Kernel Progress: %.3f%%\r", (j)/float(height) *100.0f);
		fflush(stdout);
	}



	// HitInfo* result = new HitInfo[width*height];
	// hipMemcpy(measureHIArray, dev_eyeMPs, width*height*sizeof(HitInfo), hipMemcpyDeviceToHost );
	
	



	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}
	// hipDeviceReset();
	// hipProfilerStop();
	
	return measureHIArray;
}

