#include "hip/hip_runtime.h"
#include "FinalPassKernel.h"
#include <stdio.h>

#define E (2.7182818284f)

using namespace std;


__global__
void finalPassKernel(const int height, const int width, const HitInfo* dev_scatteringMPs, const int scatteringMPsSize, const HitInfo* hiArray) {
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	// HitInfo* hi = dev_eyeMPs[j*width + i];
	HitInfo hi = hiArray[j];

	// if(hi == NULL) return;

	const float my = 1.3;
	const float sigmaS = 2.6f		* 60; // TranslucentMaterialScale;
	const float sigmaA = 0.0041f 	* 60; //TranslucentMaterialScale;
	const float sigmaT = sigmaS + sigmaA;
	// const float alpha = sigmaS / sigmaT;
	const float sigmaTR = sqrt(3.0f*sigmaA*sigmaT);
	const float lu = 1.0f/sigmaT;
	const float Fdr =  -1.440f/(my*my) + 0.710f*my + 0.668f + 0.0636f*my;
	const float Fdt = 1.0f - Fdr;
	const float A = (1 + Fdr) / (1 - Fdr);
	const float zr = lu;
	const float zv = lu*(1.0f + 4.0f/(3.0f*A));

	// const int scatteringMPsIndex = blockIdx.x*blockDim.x + threadIdx.x;
	const HitInfo sHI = dev_scatteringMPs[i];
	
	const float r2 = (hi.P - sHI.P).length2();
	const float dr = sqrt(r2+zr*zr);
	const float dv = sqrt(r2+zv*zv);
	const float C1 = zr * (sigmaTR + 1.0f/dr);
	const float C2 = zv * (sigmaTR + 1.0f/dv);

	const float dMoOverAlphaPhi = 1.0f/(4.0f*PI) * (C1*(pow(E,-sigmaTR*dr)/dr*dr) + C2*(pow(E,-sigmaTR*dv)/dv*dv));
	const Vector3 MoP = Fdt * dMoOverAlphaPhi * sHI.flux * sHI.r2 * PI;

	// scatteringMPsFlux[i] = MoP;
	hi.flux += MoP;


	
}

extern "C" __host__
HitInfo* finalPass(Image* img, HitInfo* scatteringMPs, int scatteringMPsSize, HitInfo* measureHIArray, Camera* cam) {
	int width = img->width();
	int height = img->height();

	static HitInfo *dev_scatteringMPs, *dev_eyeMPs;
	// static Vector3 *scatteringMPsFlux;
	hipMalloc((void**)&dev_scatteringMPs, scatteringMPsSize*sizeof(HitInfo));
	// hipMalloc((void**)&scatteringMPsFlux, width*height*scatteringMPsSize*sizeof(Vector3));
	hipMalloc((void**)&dev_eyeMPs, width*height*sizeof(HitInfo));
	

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	hipMemcpy( dev_scatteringMPs, scatteringMPs, scatteringMPsSize*sizeof(HitInfo), hipMemcpyHostToDevice );
	err = hipGetLastError();
	if( err != hipSuccess ) {
		printf("\nCuda error detected when copying scatteringMPs: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	hipMemcpy( dev_eyeMPs, measureHIArray, width*height*sizeof(HitInfo), hipMemcpyHostToDevice );
	err = hipGetLastError();
	if( err != hipSuccess ) {
		printf("\nCuda error detected when copying measureHIArray: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}
	// Vector3 *fluxArray = new Vector3[width*height];

	// Kernel block dimensions
	// const dim3 blockDim(8,8);

	// for (int j = 0; j < img->height(); ++j) {
	// 	for (int i = 0; i < img->width(); ++i) {
			// HitInfo hi = measureHIArray[j*width + i];
			// if(hi.t == 0.0f) continue;

			const dim3 dimBlock(16, 16);
			const dim3 dimGrid(scatteringMPsSize/dimBlock.x, width*height/dimBlock.y);

			finalPassKernel<<<dimGrid, dimBlock>>>(height, width, dev_scatteringMPs, scatteringMPsSize, dev_eyeMPs);
			// cout << "after kernel" << endl;
			// hipMemcpy(fluxArray, scatteringMPsFlux, width*height*scatteringMPsSize*sizeof(Vector3), hipMemcpyDeviceToHost );
			// Vector3 flux;
			// for(int _j = 0; _j < width*height; _j++) {
			// 	for(int _i = 1; _i < scatteringMPsSize; _i++) {
			// 		measureHIArray[_j].flux += fluxArray[_i][_j];
			// 		// fluxArray[0][_j] += fluxArray[_i][_j];
			// 		// std::cout << fluxArray[_i];
			// 	}
			// }
			// cout << "after sum" << endl;
			// cout << measureHIArray[j*width+i].flux << " ";
			// measureHIArray[j*width+i].flux = flux;
			// cout << measureHIArray[j*width+i].flux << endl;



			const char* error = hipGetErrorString(hipPeekAtLastError());
			printf("%s\n", error);
			error = hipGetErrorString(hipDeviceSynchronize());
			printf("%s\n", error);




			// Check for errors
			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
	// 	}
	// }



	// HitInfo* result = new HitInfo[width*height];
	hipMemcpy(measureHIArray, dev_eyeMPs, width*height*sizeof(HitInfo), hipMemcpyDeviceToHost );
	
	



	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected when copying result back: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	return measureHIArray;
}

