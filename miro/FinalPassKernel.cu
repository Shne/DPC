#include "hip/hip_runtime.h"
#include "FinalPassKernel.h"
#include <stdio.h>
// #include "hip/hip_runtime_api.h"

#define E (2.7182818284f)

using namespace std;

__global__
void finalPassKernel(const int height, const int width, const HitInfo* dev_scatteringMPs, const int scatteringMPsSize, const Vector3 hiP, Vector3 *scatteringMPsFlux,
                     const float sigmaTR, const float Fdt, const float zr, const float zv) {
	
	// const HitInfo hi = hiArray[hiIndex];

/*
	const float my = 1.3;
	const float sigmaS = 2.6f		* translucentMaterialScale;
	const float sigmaA = 0.0041f 	* translucentMaterialScale;
	const float sigmaT = sigmaS + sigmaA;
	// const float alpha = sigmaS / sigmaT;
	const float sigmaTR = sqrt(3.0f*sigmaA*sigmaT);
	const float lu = 1.0f/sigmaT;
	const float Fdr =  -1.440f/(my*my) + 0.710f*my + 0.668f + 0.0636f*my;
	const float Fdt = 1.0f - Fdr;
	const float A = (1 + Fdr) / (1 - Fdr);
	const float zr = lu;
	const float zv = lu*(1.0f + 4.0f/(3.0f*A));
*/

/*
	//PRECOMPUTED VALUES
	// const float my = 1.3;
	// const float sigmaS = 156;
	// const float sigmaA = 0.246;
	// const float sigmaT = 156.246;
	const float sigmaTR = 10.73822834549535922654;
	// const float lu = 0.00640016384419441138;
	// const float Fdr = 0.82160899408284023669;
	const float Fdt = 0.17839100591715976331;
	// const float A = 10.21132755386080971017;
	const float zr = 0.00640016384419441138;
	const float zv = 0.00723585849283319265;

*/


	// MULTIPLE SCATTER
	const int scatteringMPsIndex = blockIdx.x*blockDim.x + threadIdx.x;
	const HitInfo sHI = dev_scatteringMPs[scatteringMPsIndex];
	// localScatteringMPs[threadIdx.x] = dev_scatteringMPs[scatteringMPsIndex];
	// __syncthreads();
	
	const float r2 = (hiP - sHI.P).length2();
	const float dr = sqrtf(r2+zr*zr);
	const float dv = sqrtf(r2+zv*zv);
	const float C1 = zr * (sigmaTR + 1.0f/dr);
	const float C2 = zv * (sigmaTR + 1.0f/dv);

	const float dMoOverAlphaPhi = 1.0f/(4.0f*PI) * (C1*(powf(E,-sigmaTR*dr)/dr*dr) + C2*(powf(E,-sigmaTR*dv)/dv*dv));
	const Vector3 MoP = Fdt * dMoOverAlphaPhi * sHI.flux * sHI.r2 * PI;

	scatteringMPsFlux[scatteringMPsIndex] = MoP;
	// hi.flux += MoP;	
}










extern "C" __host__
HitInfo* finalPass(const int width, const int height, const HitInfo* scatteringMPs, const int scatteringMPsSize, HitInfo* measureHIArray, const float translucentMaterialScale) {
	// hipProfilerStart();

	static HitInfo *dev_scatteringMPs;//, *dev_eyeMPs;
	static Vector3 *scatteringMPsFlux;
	hipMalloc((void**)&dev_scatteringMPs, scatteringMPsSize*sizeof(HitInfo));
	hipMalloc((void**)&scatteringMPsFlux, scatteringMPsSize*sizeof(Vector3));
	// hipMalloc((void**)&dev_eyeMPs, width*height*sizeof(HitInfo));
	

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	hipMemcpy( dev_scatteringMPs, scatteringMPs, scatteringMPsSize*sizeof(HitInfo), hipMemcpyHostToDevice );
	// hipMemcpy( dev_eyeMPs, measureHIArray, width*height*sizeof(HitInfo), hipMemcpyHostToDevice );

	Vector3 *perPixelFlux = new Vector3[scatteringMPsSize];


	//VALUES FOR DIPOLE DIFFUSION MULTIPLE SCATTER
	const float my = 1.3;
	const float sigmaS = 2.6f		* translucentMaterialScale;
	const float sigmaA = 0.0041f 	* translucentMaterialScale;
	const float sigmaT = sigmaS + sigmaA;
	// const float alpha = sigmaS / sigmaT;
	const float sigmaTR = sqrt(3.0f*sigmaA*sigmaT);
	const float lu = 1.0f/sigmaT;
	const float Fdr =  -1.440f/(my*my) + 0.710f*my + 0.668f + 0.0636f*my;
	const float Fdt = 1.0f - Fdr;
	const float A = (1 + Fdr) / (1 - Fdr);
	const float zr = lu;
	const float zv = lu*(1.0f + 4.0f/(3.0f*A));

	std::cout << "scatteringMPsSize: " << scatteringMPsSize << " total size: " << scatteringMPsSize*sizeof(HitInfo) << std::endl;

	for (int j = 0; j < height; ++j) {
		for (int i = 0; i < width; ++i) {
			HitInfo hi = measureHIArray[j*width + i];
			if(hi.material == NULL) continue;

			dim3 dimBlock(64);
			dim3 dimGrid = scatteringMPsSize/dimBlock.x;

			finalPassKernel<<<dimGrid, dimBlock>>>(height, width, dev_scatteringMPs, scatteringMPsSize, hi.P, scatteringMPsFlux, sigmaTR, Fdt, zr, zv);

			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
			// cout << "after kernel" << endl;
			hipMemcpy(perPixelFlux, scatteringMPsFlux, scatteringMPsSize*sizeof(Vector3), hipMemcpyDeviceToHost );
			Vector3 flux;
			for(int _i = 0; _i < scatteringMPsSize; _i++) {
				flux += perPixelFlux[_i];
				// std::cout << perPixelFlux[_i];
			}
			// cout << "after sum" << endl;
			// cout << measureHIArray[j*width+i].flux << " ";
			measureHIArray[j*width+i].flux = flux;
			// cout << measureHIArray[j*width+i].flux << endl;

			// Check for errors
			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
		}
		printf("Kernel Progress: %.3f%%\r", (j)/float(height) *100.0f);
		fflush(stdout);
	}



	// HitInfo* result = new HitInfo[width*height];
	// hipMemcpy(measureHIArray, dev_eyeMPs, width*height*sizeof(HitInfo), hipMemcpyDeviceToHost );
	
	



	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}
	// hipDeviceReset();
	// hipProfilerStop();
	
	return measureHIArray;
}

