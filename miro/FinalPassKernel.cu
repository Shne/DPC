#include "hip/hip_runtime.h"
#include "FinalPassKernel.h"
#include <stdio.h>
// #include "hip/hip_runtime_api.h"

#define E (2.7182818284f)

using namespace std;


texture <float4> scatteringPositions_tex;
texture <float4> scatteringFlux_tex;
texture <float> scatteringR2_tex;



__global__
void finalPassKernel(const int height, const int width, /*const HitInfo* dev_scatteringMPs,*/ const int scatteringMPsSize, const Vector3 hiP, Vector3 *scatteringMPsFlux,
                     const float sigmaTR, const float Fdt, const float zr, const float zv) {
	
	// const HitInfo hi = hiArray[hiIndex];

/*
	const float my = 1.3;
	const float sigmaS = 2.6f		* translucentMaterialScale;
	const float sigmaA = 0.0041f 	* translucentMaterialScale;
	const float sigmaT = sigmaS + sigmaA;
	// const float alpha = sigmaS / sigmaT;
	const float sigmaTR = sqrt(3.0f*sigmaA*sigmaT);
	const float lu = 1.0f/sigmaT;
	const float Fdr =  -1.440f/(my*my) + 0.710f*my + 0.668f + 0.0636f*my;
	const float Fdt = 1.0f - Fdr;
	const float A = (1 + Fdr) / (1 - Fdr);
	const float zr = lu;
	const float zv = lu*(1.0f + 4.0f/(3.0f*A));
*/

/*
	//PRECOMPUTED VALUES
	// const float my = 1.3;
	// const float sigmaS = 156;
	// const float sigmaA = 0.246;
	// const float sigmaT = 156.246;
	const float sigmaTR = 10.73822834549535922654;
	// const float lu = 0.00640016384419441138;
	// const float Fdr = 0.82160899408284023669;
	const float Fdt = 0.17839100591715976331;
	// const float A = 10.21132755386080971017;
	const float zr = 0.00640016384419441138;
	const float zv = 0.00723585849283319265;

*/


	// MULTIPLE SCATTER
	const int scatteringMPsIndex = blockIdx.x*blockDim.x + threadIdx.x;
	// const HitInfo sHI = dev_scatteringMPs[scatteringMPsIndex];
	const float4 scatteringP_f = tex1Dfetch(scatteringPositions_tex, scatteringMPsIndex);
	const float4 scatteringFlux_f = tex1Dfetch(scatteringFlux_tex, scatteringMPsIndex);
	const float scatteringR2 = tex1Dfetch(scatteringR2_tex, scatteringMPsIndex);

	const Vector3 scatteringP = Vector3(scatteringP_f.x, scatteringP_f.y, scatteringP_f.z);
	const Vector3 scatteringFlux = Vector3(scatteringFlux_f.x, scatteringFlux_f.y, scatteringFlux_f.z);


	// localScatteringMPs[threadIdx.x] = dev_scatteringMPs[scatteringMPsIndex];
	// __syncthreads();
	
	const float r2 = (hiP - scatteringP).length2();
	const float dr = sqrtf(r2+zr*zr);
	const float dv = sqrtf(r2+zv*zv);
	const float C1 = zr * (sigmaTR + 1.0f/dr);
	const float C2 = zv * (sigmaTR + 1.0f/dv);

	const float dMoOverAlphaPhi = 1.0f/(4.0f*PI) * (C1*(powf(E,-sigmaTR*dr)/dr*dr) + C2*(powf(E,-sigmaTR*dv)/dv*dv));
	const Vector3 MoP = Fdt * dMoOverAlphaPhi * scatteringFlux * scatteringR2 * PI;

	scatteringMPsFlux[scatteringMPsIndex] = MoP;
	// hi.flux += MoP;	
}










extern "C" __host__
HitInfo* finalPass(const int width, const int height, const HitInfo* scatteringMPs, const int scatteringMPsSize, HitInfo* measureHIArray, const float translucentMaterialScale) {
	// hipProfilerStart();



	//split scatteringMPs into 3 arrays of position, flux and r2. to be able to use textures
	float4 scatteringPositions[scatteringMPsSize];
	float4 scatteringFlux[scatteringMPsSize];
	float scatteringR2[scatteringMPsSize];
	for(int i = 0; i<scatteringMPsSize; i++) {
		HitInfo sHI = scatteringMPs[i];
		scatteringPositions[i] = make_float4(sHI.P.x, sHI.P.y, sHI.P.z, 0.0f);
		scatteringFlux[i] = make_float4(sHI.flux.x, sHI.flux.y, sHI.flux.z, 0.0f);
		scatteringR2[i] = scatteringMPs[i].r2;
	}

	static float4 *dev_scatteringPositions, *dev_scatteringFlux;
	static float *dev_scatteringR2;
	hipMalloc((void**)&dev_scatteringPositions, scatteringMPsSize*sizeof(float4));
	hipMalloc((void**)&dev_scatteringFlux, scatteringMPsSize*sizeof(float4));
	hipMalloc((void**)&dev_scatteringR2, scatteringMPsSize*sizeof(float));

	std::cout << "allocated" << std::endl;

	hipMemcpy( dev_scatteringPositions, scatteringPositions, scatteringMPsSize*sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( dev_scatteringFlux, scatteringFlux, scatteringMPsSize*sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( dev_scatteringR2, scatteringR2, scatteringMPsSize*sizeof(float), hipMemcpyHostToDevice );

	std::cout << "Memcpy'ed" << std::endl;

	hipBindTexture( NULL, scatteringPositions_tex, dev_scatteringPositions, scatteringMPsSize*sizeof(float4));
	hipBindTexture( NULL, scatteringFlux_tex, dev_scatteringFlux, scatteringMPsSize*sizeof(float4));
	hipBindTexture( NULL, scatteringR2_tex, dev_scatteringR2, scatteringMPsSize*sizeof(float));

	std::cout << "Textures bound" << std::endl;

	// static HitInfo *dev_scatteringMPs;//, *dev_eyeMPs;
	static Vector3 *scatteringMPsFlux;
	// hipMalloc((void**)&dev_scatteringMPs, scatteringMPsSize*sizeof(HitInfo));
	hipMalloc((void**)&scatteringMPsFlux, scatteringMPsSize*sizeof(Vector3));
	// hipMalloc((void**)&dev_eyeMPs, width*height*sizeof(HitInfo));
	

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	// hipMemcpy( dev_scatteringMPs, scatteringMPs, scatteringMPsSize*sizeof(HitInfo), hipMemcpyHostToDevice );
	// hipMemcpy( dev_eyeMPs, measureHIArray, width*height*sizeof(HitInfo), hipMemcpyHostToDevice );

	Vector3 *perPixelFlux = new Vector3[scatteringMPsSize];


	//VALUES FOR DIPOLE DIFFUSION MULTIPLE SCATTER
	const float my = 1.3;
	const float sigmaS = 2.6f		* translucentMaterialScale;
	const float sigmaA = 0.0041f 	* translucentMaterialScale;
	const float sigmaT = sigmaS + sigmaA;
	// const float alpha = sigmaS / sigmaT;
	const float sigmaTR = sqrt(3.0f*sigmaA*sigmaT);
	const float lu = 1.0f/sigmaT;
	const float Fdr =  -1.440f/(my*my) + 0.710f*my + 0.668f + 0.0636f*my;
	const float Fdt = 1.0f - Fdr;
	const float A = (1 + Fdr) / (1 - Fdr);
	const float zr = lu;
	const float zv = lu*(1.0f + 4.0f/(3.0f*A));

	std::cout << "scatteringMPsSize: " << scatteringMPsSize << " total size: " << scatteringMPsSize*sizeof(HitInfo) << std::endl;

	for (int j = 0; j < height; ++j) {
		for (int i = 0; i < width; ++i) {
			HitInfo hi = measureHIArray[j*width + i];
			if(hi.material == NULL) continue;

			dim3 dimBlock(64);
			dim3 dimGrid = scatteringMPsSize/dimBlock.x;

			finalPassKernel<<<dimGrid, dimBlock>>>(height, width, /*dev_scatteringMPs,*/ scatteringMPsSize, hi.P, scatteringMPsFlux, sigmaTR, Fdt, zr, zv);

			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
			// cout << "after kernel" << endl;
			hipMemcpy(perPixelFlux, scatteringMPsFlux, scatteringMPsSize*sizeof(Vector3), hipMemcpyDeviceToHost );
			Vector3 flux;
			for(int _i = 0; _i < scatteringMPsSize; _i++) {
				flux += perPixelFlux[_i];
				// std::cout << perPixelFlux[_i];
			}
			// cout << "after sum" << endl;
			// cout << measureHIArray[j*width+i].flux << " ";
			measureHIArray[j*width+i].flux = flux;
			// cout << measureHIArray[j*width+i].flux << endl;

			// Check for errors
			err = hipGetLastError();
			if( err != hipSuccess ) {
				printf("\nCuda error detected in 'finalPassKernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
				exit(1);
			}
		}
		printf("Kernel Progress: %.3f%%\r", (j)/float(height) *100.0f);
		fflush(stdout);
	}



	// HitInfo* result = new HitInfo[width*height];
	// hipMemcpy(measureHIArray, dev_eyeMPs, width*height*sizeof(HitInfo), hipMemcpyDeviceToHost );
	
	



	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}
	// hipDeviceReset();
	// hipProfilerStop();

	hipFree(scatteringMPsFlux);
	// hipFree(dev_scatteringMPs);

	hipUnbindTexture(scatteringPositions_tex);
	hipUnbindTexture(scatteringFlux_tex);
	hipUnbindTexture(scatteringR2_tex);
	
	return measureHIArray;
}

